
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <iostream>


// CUDA kernel. Each thread takes care of one element of c
__global__ void deviceIteration(int *c, int *a, int H, int W)
{
    // Tomamos el indice y lugar donde se calculará si vive o no.
    const int k = blockIdx.x * blockDim.x + threadIdx.x;

    // En caso de pasarnos no lo consideramos.
    if (k < W*H) {

        // Calculamos la posición en la matriz.
        const int i = k / H;
        const int j = k % H;

        int sum = 0;

        // Ahora obtenemos los indices de los vecinos.
        const int left = (i + W - 1) % W;
        const int right = (i + 1) % W;
        const int down = (j + 1) % H;
        const int up = (j + H - 1) % H;

        // Calculamos la suma de los valores vecinos.

        // left
        if (a[right * H + j])
            sum++;
        // right
        if (a[left * H + j])
            sum++;
        // up
        if (a[i * H + up])
            sum++;
        // down
        if (a[i * H + down])
            sum++;
        // upright
        if (a[right * H + up])
            sum++;
        // downright
        if (a[right * H + down])
            sum++;
        // upleft
        if (a[left * H + up])
            sum++;
        // downleft
        if (a[left * H + down])
            sum++;

        int value = a[k];
        int result = 0;
        // Guardamos el resultado obtenido, si esta vivo o no
        if ((value == 1 && (sum == 2 || sum == 3)) || (value == 0 && (sum == 3 || sum == 6))) {
            result = 1;
        }

        // Lo dejamos en la matriz final.
        c[k] = result;
    }
}
// CUDA kernel. Each thread takes care of one element of c
__global__ void deviceIterationNotIf(int *c, int *a, int H, int W)
{
    // Tomamos el indice y lugar donde se calculará si vive o no.
    const int k = blockIdx.x * blockDim.x + threadIdx.x;
    
    // En caso de pasarnos no lo consideramos.
    if (k < W*H) {

        // Calculamos la posición en la matriz.
        const int i = k / H;
        const int j = k % H;

        int sum = 0;

        // Ahora obtenemos los indices de los vecinos.
        const int left = (i + W - 1) % W;
        const int right = (i + 1) % W;
        const int down = (j + 1) % H;
        const int up = (j + H - 1) % H;

        // Calculamos la suma de los valores vecinos.

        // left
        sum += a[right * H + j];
        // right
        sum += a[left * H + j];
        // up
        sum += a[i * H + up];
        // down
        sum += a[i * H + down];
        // upright
        sum += a[right * H + up];
        // downright
        sum += a[right * H + down];
        // upleft
        sum += a[left * H + up];
        // downleft
        sum += a[left * H + down];

        int value = a[k];
        // Guardamos el resultado obtenido, si esta vivo o no
        c[k] = (value == 1 && (sum == 2 || sum == 3)) || (value == 0 && (sum == 3 || sum == 6));
    }
}


void run(int* a, int W, int H){


    //Host output vector
    int *h_c;
    // Device input vectors
    int *d_a;
    //Device output vector
    int *d_c;

    // Size, in bytes, of each vector
    size_t bytes = W*H*sizeof(int);

    // Allocate memory for each vector on host
    h_c = (int*)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc((void **) &d_a, bytes);
    hipMalloc((void **) &d_c, bytes);

    // Copy host vectors to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize, n;

    // Tamaño de la matriz.
    n = H*W;

    // Tamaño del bloque. Elegir entre 32 y 31.
    //blockSize = 32;
    blockSize = 8;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    deviceIterationNotIf<<< gridSize, blockSize >>>(d_c, d_a, H, W);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    int i,j;
    for(i=0; i<W; i++) {
        for(j=0; j<H; j++) {
            a[i*H + j] = h_c[i*H + j];
        }
    }

    // Release device memory
    hipFree(d_a);
    hipFree(d_c);

    // Release host memory
    free(h_c);
}


