
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <iostream>


// CUDA kernel. Each thread takes care of one element of c
__global__ void deviceIteration(int *c, int *a, int H, int W)
{
    // Get our global thread ID
    const int k = blockIdx.x;
    if (k < W*H) {

        const int i = k / H;
        const int j = k % H;

        int sum = 0;

        const int left = (i + W - 1) % W;
        const int right = (i + 1) % W;
        const int down = (j + 1) % H;
        const int up = (j + H - 1) % H;

        // left
        sum += a[right * H + j];
        // right
        sum += a[left * H + j];
        // up
        sum += a[i * H + up];
        // down
        sum += a[i * H + down];
        // upright
        sum += a[right * H + up];
        // downright
        sum += a[right * H + down];
        // upleft
        sum += a[left * H + up];
        // downleft
        sum += a[left * H + down];

        int value = a[k];
        int result = 0;
        if ((value == 1 && (sum == 2 || sum == 3)) || (value == 0 && (sum == 3 || sum == 6))) {
            result = 1;
        }

        c[k] = result;
    }
}


void run(int* a, int W, int H){


    //Host output vector
    int *h_c;

    // Device input vectors
    int *d_a;
    //Device output vector
    int *d_c;

    // Size, in bytes, of each vector
    size_t bytes = W*H*sizeof(int);

    // Allocate memory for each vector on host
    h_c = (int*)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc((void **) &d_a, bytes);
    hipMalloc((void **) &d_c, bytes);

    // Copy host vectors to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);

    //int blockSize, gridSize;

    // Execute the kernel
    deviceIteration<<<W*H, 1>>>(d_c, d_a, H, W);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    int i,j;
    for(i=0; i<W; i++) {
        for(j=0; j<H; j++) {
            a[i*H + j] = h_c[i*H + j];
        }
    }

    // Release device memory
    hipFree(d_a);
    hipFree(d_c);

    // Release host memory
    free(h_c);
}


